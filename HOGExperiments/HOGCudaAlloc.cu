#include "HOGCudaAlloc.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void cudaHOGAlloc( float **desc, int descSize, float **hog, int hogSize, float **mag, float **grad, int magGradSize ){
	hipMalloc( (void**)desc, descSize );
	hipMalloc( (void**)hog, hogSize );
	hipMalloc( mag, magGradSize );
	hipMalloc( grad, magGradSize );
	hipMemset( mag, 0, magGradSize );
	hipMemset( grad, 0, magGradSize );	
}

void cudaHOGFree( float **desc, float **hog, float **mag, float **grad ){
	hipFree( *desc );
	hipFree( *hog );
	hipFree( *mag );
	hipFree( *grad );
}
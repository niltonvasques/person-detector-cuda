#include "hip/hip_runtime.h"
#include "FastHOGSVM.h"
#include "FastHOGUtils.h"

texture<float, 1, hipReadModeElementType> texSVM;
hipArray *svmArray = 0;

hipChannelFormatDesc channelDescSVM;

//extern int scaleCount;
//extern int hNumberOfWindowsX, hNumberOfWindowsY;
//extern int hNumberOfBlockPerWindowX, hNumberOfBlockPerWindowY;
//extern int rNumberOfWindowsX, rNumberOfWindowsY;

extern __shared__ float1 allSharedF1[];

float svmBias;

__host__ void InitSVM(float _svmBias, float* svmWeights, int svmWeightsCount)
{
	channelDescSVM = hipCreateChannelDesc<float>();
	cutilSafeCall(hipMallocArray(&svmArray, &channelDescSVM, svmWeightsCount, 1));
	cutilSafeCall(hipMemcpyToArray(svmArray, 0, 0, svmWeights, svmWeightsCount * sizeof(float), hipMemcpyHostToDevice));
	svmBias = _svmBias;
}

__host__ void CloseSVM()
{
	cutilSafeCall(hipFreeArray(svmArray));
}

__global__ void linearSVMEvaluation(float1* svmScores, float svmBias,
									float1* blockHistograms, int noHistogramBins,
									int windowSizeX, int windowSizeY, int hogBlockCountX, int hogBlockCountY,
									int cellSizeX, int cellSizeY,
									int numberOfBlockPerWindowX, int numberOfBlockPerWindowY,
									int blockSizeX, int blockSizeY,
									int alignedBlockDimX,
									int scaleId, int scaleCount,
									int hNumberOfWindowsX, int hNumberOfWindowsY,
									int width, int height)
{
	int i;
	int texPos;
	float1 localValue;
	float texValue;

	float1* smem = (float1*) allSharedF1;

	int gmemPosWindow, gmemPosInWindow, gmemPosInWindowDown, smemLocalPos, smemTargetPos;
	int gmemStride = hogBlockCountX * noHistogramBins * blockSizeX;

	gmemPosWindow = blockIdx.x * noHistogramBins * blockSizeX + blockIdx.y * blockSizeY * gmemStride;
	gmemPosInWindow = gmemPosWindow + threadIdx.x;
	smemLocalPos = threadIdx.x;

	int val1 = (blockSizeY * blockSizeX * noHistogramBins) * numberOfBlockPerWindowY;
	int val2 = blockSizeX * noHistogramBins;
	localValue.x = 0;

	if (blockIdx.x == 10 && blockIdx.y == 8)
	{
		int asasasa;
		asasasa = 0;
		asasasa++;
	}

	for (i = 0; i<blockSizeY * numberOfBlockPerWindowY; i++)
	{
		gmemPosInWindowDown = gmemPosInWindow + i * gmemStride;
		texPos = threadIdx.x % val2 + i * val2 + threadIdx.x / val2 * val1;
		texValue =  tex1D(texSVM, texPos);
		localValue.x += blockHistograms[gmemPosInWindowDown].x * texValue;
	}

	smem[smemLocalPos] = localValue;

	__syncthreads();

	for(unsigned int s = alignedBlockDimX >> 1; s>0; s>>=1)
	{
		if (threadIdx.x < s && (threadIdx.x + s) < blockDim.x)
		{
			smemTargetPos = threadIdx.x + s;
			smem[smemLocalPos].x += smem[smemTargetPos].x;
		}

		__syncthreads();
	}

	if (threadIdx.x == 0)
	{
		smem[smemLocalPos].x -= svmBias;
		svmScores[blockIdx.x + blockIdx.y * hNumberOfWindowsX + scaleId * hNumberOfWindowsX * hNumberOfWindowsY] = smem[smemLocalPos];
	}

	if (blockIdx.x == 10 && blockIdx.y == 8)
	{
		int asasasa;
		asasasa = 0;
		asasasa++;
	}
}

__host__ void ResetSVMScores(FastHOG_::FastHOG *fHOG, float1* svmScores)
{
	cutilSafeCall(hipMemset(svmScores, 0, sizeof(float) * fHOG->scaleCount * fHOG->hNumberOfWindowsX * fHOG->hNumberOfWindowsY));
}

__host__ void LinearSVMEvaluation(FastHOG_::FastHOG *fHOG, float1* svmScores, float1* blockHistograms, int noHistogramBins,
								  int windowSizeX, int windowSizeY,
								  int cellSizeX, int cellSizeY, int blockSizeX, int blockSizeY,
								  int hogBlockCountX, int hogBlockCountY,
								  int scaleId, int width, int height)
{
	int rNumberOfWindowsX = (width-windowSizeX)/cellSizeX + 1;
	int rNumberOfWindowsY = (height-windowSizeY)/cellSizeY + 1;

	dim3 threadCount = dim3(noHistogramBins * blockSizeX * fHOG->hNumberOfBlockPerWindowX);
	dim3 blockCount = dim3(rNumberOfWindowsX, rNumberOfWindowsY);

	int alignedBlockDimX = iClosestPowerOfTwo(noHistogramBins * blockSizeX * fHOG->hNumberOfBlockPerWindowX);

	cutilSafeCall(hipBindTextureToArray(texSVM, svmArray, channelDescSVM));

	linearSVMEvaluation<<<blockCount, threadCount, noHistogramBins * blockSizeX * fHOG->hNumberOfBlockPerWindowX * sizeof(float1)>>>
		(svmScores, svmBias, blockHistograms, noHistogramBins,
		windowSizeX, windowSizeY, hogBlockCountX, hogBlockCountY, cellSizeX, cellSizeY,
		fHOG->hNumberOfBlockPerWindowX, fHOG->hNumberOfBlockPerWindowY,
		blockSizeX, blockSizeY, alignedBlockDimX, scaleId, fHOG->scaleCount,
		fHOG->hNumberOfWindowsX, fHOG->hNumberOfWindowsY, width, height);

	cutilSafeCall(hipUnbindTexture(texSVM));
}
